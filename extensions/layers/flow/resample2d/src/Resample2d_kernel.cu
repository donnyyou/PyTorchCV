#include "hip/hip_runtime.h"
#include <THC.h>
#include <THCGeneral.h>
#include <stdbool.h>
#include <stdio.h>

#define CUDA_NUM_THREADS 512 
#define THREADS_PER_BLOCK 64 

#define DIM0(TENSOR) ((TENSOR).x)
#define DIM1(TENSOR) ((TENSOR).y)
#define DIM2(TENSOR) ((TENSOR).z)
#define DIM3(TENSOR) ((TENSOR).w)

#define DIM3_INDEX(TENSOR, xx, yy, zz, ww) ((TENSOR)[((xx) * (TENSOR##_stride.x)) + ((yy) * (TENSOR##_stride.y)) + ((zz) * (TENSOR##_stride.z)) + ((ww) * (TENSOR##_stride.w))])

#ifdef __cplusplus
    extern "C" {
#endif

__global__ void kernel_Resample2d_updateOutput(const int n, const float* input1, const long4 input1_size, const long4 input1_stride,
    const float* input2, const long4 input2_size, const long4 input2_stride, float* output, const long4 output_size, const long4 output_stride, int kernel_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    float val = 0.0;

    int dim_b = DIM0(output_size);
    int dim_c = DIM1(output_size);
    int dim_h = DIM2(output_size);
    int dim_w = DIM3(output_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int c = ( index / dim_hw )  % dim_c;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    float dx = DIM3_INDEX(input2, b, 0, y, x);
    float dy = DIM3_INDEX(input2, b, 1, y, x);

    float xf = float(x) + dx;
    float yf = float(y) + dy;
    float alpha = xf - floor(xf); // alpha
    float beta = yf - floor(yf); // beta

    int xL = max(min( int (floor(xf)),    dim_w-1), 0);
    int xR = max(min( int (floor(xf)+1), dim_w -1), 0);
    int yT = max(min( int (floor(yf)),    dim_h-1), 0);
    int yB = max(min( int (floor(yf)+1),  dim_h-1), 0);

    for (int fy = 0; fy < kernel_size; fy += 1) {
        for (int fx = 0; fx < kernel_size; fx += 1) {
            val += (1. - alpha)*(1. - beta) * DIM3_INDEX(input1, b, c, yT + fy, xL + fx);
            val +=    (alpha)*(1. - beta) * DIM3_INDEX(input1, b, c, yT + fy, xR + fx);
            val +=    (1. - alpha)*(beta) * DIM3_INDEX(input1, b, c, yB + fy, xL + fx);
            val +=       (alpha)*(beta) * DIM3_INDEX(input1, b, c, yB + fy, xR + fx);
        }
    }

    output[index] = val;

}


__global__ void kernel_Resample2d_backward_input1(
    const int n, const float* input1, const long4 input1_size, const long4 input1_stride, const float* input2, const long4 input2_size, const long4 input2_stride,
    const float* gradOutput, const long4 gradOutput_size, const long4 gradOutput_stride, float* gradInput, const long4 gradInput_size, const long4 gradInput_stride, int kernel_size) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    int dim_b = DIM0(gradOutput_size);
    int dim_c = DIM1(gradOutput_size);
    int dim_h = DIM2(gradOutput_size);
    int dim_w = DIM3(gradOutput_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int c = ( index / dim_hw )  % dim_c;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    float dx = DIM3_INDEX(input2, b, 0, y, x);
    float dy = DIM3_INDEX(input2, b, 1, y, x);

    float xf = float(x) + dx;
    float yf = float(y) + dy;
    float alpha = xf - int(xf); // alpha
    float beta = yf - int(yf); // beta

    int idim_h = DIM2(input1_size);
    int idim_w = DIM3(input1_size);

    int xL = max(min( int (floor(xf)),    idim_w-1), 0);
    int xR = max(min( int (floor(xf)+1), idim_w -1), 0);
    int yT = max(min( int (floor(yf)),    idim_h-1), 0);
    int yB = max(min( int (floor(yf)+1),  idim_h-1), 0);

    for (int fy = 0; fy < kernel_size; fy += 1) {
        for (int fx = 0; fx < kernel_size; fx += 1) {
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yT + fy), (xL + fx)), (1-alpha)*(1-beta) * DIM3_INDEX(gradOutput, b, c, y, x));
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yT + fy), (xR + fx)),   (alpha)*(1-beta) * DIM3_INDEX(gradOutput, b, c, y, x));
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yB + fy), (xL + fx)),   (1-alpha)*(beta) * DIM3_INDEX(gradOutput, b, c, y, x));
            atomicAdd(&DIM3_INDEX(gradInput, b, c, (yB + fy), (xR + fx)),     (alpha)*(beta) * DIM3_INDEX(gradOutput, b, c, y, x));
        }
    }

}

__global__ void kernel_Resample2d_backward_input2(
    const int n, const float* input1, const long4 input1_size, const long4 input1_stride, const float* input2, const long4 input2_size, const long4 input2_stride,
    const float* gradOutput, const long4 gradOutput_size, const long4 gradOutput_stride, float* gradInput, const long4 gradInput_size, const long4 gradInput_stride, int kernel_size) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n) {
        return;
    }

    float output = 0.0;
    int kernel_rad = (kernel_size - 1)/2;

    int dim_b = DIM0(gradInput_size);
    int dim_c = DIM1(gradInput_size);
    int dim_h = DIM2(gradInput_size);
    int dim_w = DIM3(gradInput_size);
    int dim_chw = dim_c * dim_h * dim_w;
    int dim_hw  = dim_h * dim_w;

    int b = ( index / dim_chw ) % dim_b;
    int c = ( index / dim_hw )  % dim_c;
    int y = ( index / dim_w )   % dim_h;
    int x = ( index          )  % dim_w;

    int odim_c = DIM1(gradOutput_size);

    float dx = DIM3_INDEX(input2, b, 0, y, x);
    float dy = DIM3_INDEX(input2, b, 1, y, x);

    float xf = float(x) + dx;
    float yf = float(y) + dy;

    int xL = max(min( int (floor(xf)),    dim_w-1), 0);
    int xR = max(min( int (floor(xf)+1), dim_w -1), 0);
    int yT = max(min( int (floor(yf)),    dim_h-1), 0);
    int yB = max(min( int (floor(yf)+1),  dim_h-1), 0);
    
    if (c % 2) {
        float gamma = 1 - (xf - floor(xf)); // alpha
        for (int i = 0; i <= 2*kernel_rad; ++i) {
            for (int j = 0; j <= 2*kernel_rad; ++j) {
                for (int ch = 0; ch < odim_c; ++ch) {
                    output += (gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yB + j), (xL + i));
                    output -= (gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yT + j), (xL + i));
                    output += (1-gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yB + j), (xR + i));
                    output -= (1-gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yT + j), (xR + i));
                }
            }
        }
    }
    else {
        float gamma = 1 - (yf - floor(yf)); // alpha
        for (int i = 0; i <= 2*kernel_rad; ++i) {
            for (int j = 0; j <= 2*kernel_rad; ++j) {
                for (int ch = 0; ch < odim_c; ++ch) {
                    output += (gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yT + j), (xR + i));
                    output -= (gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yT + j), (xL + i));
                    output += (1-gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yB + j), (xR + i));
                    output -= (1-gamma) * DIM3_INDEX(gradOutput, b, ch, y, x) * DIM3_INDEX(input1, b, ch, (yB + j), (xL + i));
                }
            }
        }

    }

    gradInput[index] = output;

}

void Resample2d_kernel_forward(THCState* state, THCudaTensor* input1, THCudaTensor* input2, THCudaTensor* output, int kernel_size) {
    int n = 0;

    const long4 input1_size = make_long4(input1->size[0], input1->size[1], input1->size[2], input1->size[3]);
    const long4 input1_stride = make_long4(input1->stride[0], input1->stride[1], input1->stride[2], input1->stride[3]);

    const long4 input2_size = make_long4(input2->size[0], input2->size[1], input2->size[2], input2->size[3]);
    const long4 input2_stride = make_long4(input2->stride[0], input2->stride[1], input2->stride[2], input2->stride[3]);

    const long4 output_size = make_long4(output->size[0], output->size[1], output->size[2], output->size[3]);
    const long4 output_stride = make_long4(output->stride[0], output->stride[1], output->stride[2], output->stride[3]);

    n = THCudaTensor_nElement(state, output);
    kernel_Resample2d_updateOutput<<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
        n, THCudaTensor_data(state, input1), input1_size, input1_stride, THCudaTensor_data(state, input2), input2_size, input2_stride,
        THCudaTensor_data(state, output), output_size, output_stride, kernel_size);

    THCudaCheck(hipGetLastError());
}

void Resample2d_kernel_backward(THCState* state, THCudaTensor* input1, THCudaTensor* input2, THCudaTensor* gradOutput, THCudaTensor* gradInput1, THCudaTensor* gradInput2, int kernel_size) {
    int n = 0;

    const long4 input1_size = make_long4(input1->size[0], input1->size[1], input1->size[2], input1->size[3]);
    const long4 input1_stride = make_long4(input1->stride[0], input1->stride[1], input1->stride[2], input1->stride[3]);

    const long4 input2_size = make_long4(input2->size[0], input2->size[1], input2->size[2], input2->size[3]);
    const long4 input2_stride = make_long4(input2->stride[0], input2->stride[1], input2->stride[2], input2->stride[3]);

    const long4 gradOutput_size = make_long4(gradOutput->size[0], gradOutput->size[1], gradOutput->size[2], gradOutput->size[3]);
    const long4 gradOutput_stride = make_long4(gradOutput->stride[0], gradOutput->stride[1], gradOutput->stride[2], gradOutput->stride[3]);

    const long4 gradInput1_size = make_long4(gradInput1->size[0], gradInput1->size[1], gradInput1->size[2], gradInput1->size[3]);
    const long4 gradInput1_stride = make_long4(gradInput1->stride[0], gradInput1->stride[1], gradInput1->stride[2], gradInput1->stride[3]);

    n = THCudaTensor_nElement(state, gradOutput);
    kernel_Resample2d_backward_input1<<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
        n, THCudaTensor_data(state, input1), input1_size, input1_stride, THCudaTensor_data(state, input2), input2_size, input2_stride,
        THCudaTensor_data(state, gradOutput), gradOutput_size, gradOutput_stride, THCudaTensor_data(state, gradInput1), gradInput1_size, gradInput1_stride, kernel_size
    );

    const long4 gradInput2_size = make_long4(gradInput2->size[0], gradInput2->size[1], gradInput2->size[2], gradInput2->size[3]);
    const long4 gradInput2_stride = make_long4(gradInput2->stride[0], gradInput2->stride[1], gradInput2->stride[2], gradInput2->stride[3]);

    n = THCudaTensor_nElement(state, gradInput2);
    kernel_Resample2d_backward_input2<<< (n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>(
        n, THCudaTensor_data(state, input1), input1_size, input1_stride, THCudaTensor_data(state, input2), input2_size, input2_stride,
        THCudaTensor_data(state, gradOutput), gradOutput_size, gradOutput_stride, THCudaTensor_data(state, gradInput2), gradInput2_size, gradInput2_stride, kernel_size
    );
    THCudaCheck(hipGetLastError());
}

#ifdef __cplusplus
    }
#endif